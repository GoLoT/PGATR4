//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__
void box_filter(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  // NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
  // los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Reservar memoria para el filtro en GPU: d_filter, la cual ya esta declarada
  // Copiar el filtro  (h_filter) a memoria global de la GPU (d_filter)
}


void create_filter(float **h_filter, int *filterWidth){

  const int KernelWidth = 5; //OJO CON EL TAMA�O DEL FILTRO//
  *filterWidth = KernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[KernelWidth * KernelWidth];
  
  /*
  //Filtro gaussiano: blur
  const float KernelSigma = 2.;

  float filterSum = 0.f; //for normalization

  for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
    for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
      float filterValue = expf( -(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
      (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] = filterValue;
      filterSum += filterValue;
    }
  }

  float normalizationFactor = 1.f / filterSum;

  for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
    for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
      (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] *= normalizationFactor;
    }
  }
  */

  //Laplaciano 5x5
  (*h_filter)[0] = 0;   (*h_filter)[1] = 0;    (*h_filter)[2] = -1.;  (*h_filter)[3] = 0;    (*h_filter)[4] = 0;
  (*h_filter)[5] = 1.;  (*h_filter)[6] = -1.;  (*h_filter)[7] = -2.;  (*h_filter)[8] = -1.;  (*h_filter)[9] = 0;
  (*h_filter)[10] = -1.;(*h_filter)[11] = -2.; (*h_filter)[12] = 17.; (*h_filter)[13] = -2.; (*h_filter)[14] = -1.;
  (*h_filter)[15] = 1.; (*h_filter)[16] = -1.; (*h_filter)[17] = -2.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0;
  (*h_filter)[20] = 1.;  (*h_filter)[21] = 0;   (*h_filter)[22] = -1.; (*h_filter)[23] = 0;   (*h_filter)[24] = 0;
  
  //TODO: crear los filtros segun necesidad
  //NOTA: cuidado al establecer el tama�o del filtro a utilizar

}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redFiltered, 
                        unsigned char *d_greenFiltered, 
                        unsigned char *d_blueFiltered,
                        const int filterWidth)
{
  //TODO: Calcular tama�os de bloque
  const dim3 blockSize;
  const dim3 gridSize;

  //TODO: Lanzar kernel para separar imagenes RGBA en diferentes colores

  //TODO: Ejecutar convoluci�n. Una por canal
  

  // Recombining the results. 
  recombineChannels<<<gridSize, blockSize>>>(d_redFiltered,
                                             d_greenFiltered,
                                             d_blueFiltered,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}
