//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

#define GAUSSIAN_SZ 9
// 1 = Laplacian5x5 ; 2 = Nitidez5x5; 3 = PasoAlto5x5; 4 = Media3x3 ; 5 = Blur3x3 ; 6 = Blur5x5 ; 7 = GaussianBlur ; 8 = SobelHori3x3 ; 9 = SobelVert3x3
#define FILTER 1
//Definimos tama�o de bloque en preprocesador para facilidad al hacer pruebas
#define BLOCK_SZ 32
//Definimos tama�o de convoluci�n en preprocesador para poder inicializar array de memoria constante
#if FILTER == 4 || FILTER == 5 || FILTER == 8 || FILTER == 9
#define KERNEL_SZ 3
#elif FILTER == 7
#ifndef GAUSSIAN_SZ
#define KERNEL_SZ 3
#else
#define KERNEL_SZ GAUSSIAN_SZ
#endif
#else
#define KERNEL_SZ 5
#endif
__constant__ float d_filterConst[KERNEL_SZ*KERNEL_SZ];
//Definimos para facilitar el cambio entre los kernels de memoria compartida y global
#define SHARED 1

__global__
void box_filter_shared(const unsigned char* const inputChannel,
  unsigned char* const outputChannel,
  int numRows, int numCols,
  const float* const filter, const int filterWidth)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  // NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
  // los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.

  extern __shared__ unsigned char image_shared[];

  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
    blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  //Shared size siempre deber�a ser par, ya que blockdim.x, blockdim.y
  //y filterwidth-1 siempre deber�n ser pares
  const int sharedSize = (blockDim.x + filterWidth - 1) * (blockDim.y + filterWidth - 1);
  const int halfFilterWidth = filterWidth / 2;
  const int threadNum = threadIdx.y * blockDim.x + threadIdx.x;
  const int width = blockDim.x + filterWidth - 1;

  const int numThreads = blockDim.x * blockDim.y;
  int workingThreads, offset = 0;

  //Calculamos coordenadas de imagen de la secci�n a mapear en shared memory
  const int startX = blockIdx.x * blockDim.x - halfFilterWidth;
  const int startY = blockIdx.y * blockDim.y - halfFilterWidth;

  while(offset < sharedSize)
  {
    workingThreads = sharedSize - offset;
    workingThreads = numThreads > workingThreads ? workingThreads : numThreads;

    if(threadNum < workingThreads)
    {
      //Calculamos las coordenadas en shared memory
      int sharedY = (threadNum+offset) / width;
      int sharedX = (threadNum+offset) - sharedY * width;
      //Pasamos a coordenadas de imagen
      int imgX = sharedX + startX;
      int imgY = sharedY + startY;
      //Hacemos clamp para asegurar que no nos salimos de la imagen
      imgY = imgY >= numRows ? numRows - 1 : imgY < 0 ? 0 : imgY;
      imgX = imgX >= numCols ? numCols - 1 : imgX < 0 ? 0 : imgX;

      image_shared[threadNum + offset] = inputChannel[imgY * numCols + imgX];
    }
    offset += workingThreads;
  }

  __syncthreads();

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;
  int filterRadius = filterWidth / 2;
  float result = 0;
  for (int j = -filterRadius; j <= filterRadius; j++)
    for (int i = -filterRadius; i <= filterRadius; i++) {
      int x = threadIdx.x + halfFilterWidth + i;
      int y = threadIdx.y + halfFilterWidth + j;

      result += (float)d_filterConst[(j + filterRadius)*filterWidth + i + filterRadius] * (float)image_shared[y*width + x];
    }
  outputChannel[thread_1D_pos] = result > 255 ? 255 : result < 0 ? 0 : (char)result;
}

__global__
void box_filter(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  // NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
  // los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.

  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
    blockIdx.y * blockDim.y + threadIdx.y);
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;
  int filterRadius = filterWidth / 2;
  float result = 0;
  for (int j = -filterRadius; j <= filterRadius; j++)
    for (int i = -filterRadius; i <= filterRadius; i++)  {
      int x = thread_2D_pos.x + i;
      x = x >= numCols ? numCols - 1 : x;
      x = x < 0 ? 0 : x;
      int y = thread_2D_pos.y + j;
      y = y >= numRows ? numRows - 1 : y;
      y = y < 0 ? 0 : y;
      //Sin memoria de constantes
      //result += (float) filter[(j + filterRadius)*filterWidth + i + filterRadius] * (float) inputChannel[y*numCols + x];
      //Con memoria de constantes
      result += (float)d_filterConst[(j + filterRadius)*filterWidth + i + filterRadius] * (float)inputChannel[y*numCols + x];
    }
  outputChannel[thread_2D_pos.y * numCols + thread_2D_pos.x] = result>255?255:result<0?0:(char)result;
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Reservar memoria para el filtro en GPU: d_filter, la cual ya esta declarada
  // Copiar el filtro  (h_filter) a memoria global de la GPU (d_filter)
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_filterConst), h_filter, sizeof(float) * filterWidth * filterWidth, 0, hipMemcpyHostToDevice));
}


void create_filter(float **h_filter, int *filterWidth){

  const int KernelWidth = KERNEL_SZ; //OJO CON EL TAMA�O DEL FILTRO//
  *filterWidth = KernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[KernelWidth * KernelWidth];
  
  /*
  //Filtro gaussiano: blur
  const float KernelSigma = 2.;

  float filterSum = 0.f; //for normalization

  for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
    for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
      float filterValue = expf( -(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
      (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] = filterValue;
      filterSum += filterValue;
    }
  }

  float normalizationFactor = 1.f / filterSum;

  for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
    for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
      (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] *= normalizationFactor;
    }
  }
  */

#if FILTER == 2
  //Nitidez 5x5
  (*h_filter)[0] = -1.;     (*h_filter)[1] = -3.;   (*h_filter)[2] = -4.;   (*h_filter)[3] = -3.;     (*h_filter)[4] = -1.;
  (*h_filter)[5] = -3.;     (*h_filter)[6] = 0;     (*h_filter)[7] = 6.;    (*h_filter)[8] = 0;       (*h_filter)[9] = 3.;
  (*h_filter)[10] = -4.;    (*h_filter)[11] = 6.;   (*h_filter)[12] = 21.;  (*h_filter)[13] = 6.;     (*h_filter)[14] = -4.;
  (*h_filter)[15] = -3.;    (*h_filter)[16] = 0;    (*h_filter)[17] = 6.;   (*h_filter)[18] = 0;      (*h_filter)[19] = -3.;
  (*h_filter)[20] = -1.;    (*h_filter)[21] = -3.;  (*h_filter)[22] = -4.;  (*h_filter)[23] = -3.;    (*h_filter)[24] = -1.;

#elif FILTER == 3
  //PasoAlto 5x5
  (*h_filter)[0] = 1.;   (*h_filter)[1] = 1.;   (*h_filter)[2] = 1.;    (*h_filter)[3] = 1.;    (*h_filter)[4] = 1.;
  (*h_filter)[5] = 1.;   (*h_filter)[6] = 4.;   (*h_filter)[7] = 4.;    (*h_filter)[8] = 4.;    (*h_filter)[9] = 1.;
  (*h_filter)[10] = 1.;  (*h_filter)[11] = 4.;  (*h_filter)[12] = 12.;  (*h_filter)[13] = 4.;   (*h_filter)[14] = 1.;
  (*h_filter)[15] = 1.;  (*h_filter)[16] = 4.;  (*h_filter)[17] = 4.;   (*h_filter)[18] = 4.;   (*h_filter)[19] = 1.;
  (*h_filter)[20] = 1.;  (*h_filter)[21] = 1.;  (*h_filter)[22] = 1.;   (*h_filter)[23] = 1.;   (*h_filter)[24] = 1.;

  for (int i = 0; i < 25; i++)
    (*h_filter)[i] /= 62.0;

#elif FILTER == 4
  //Media3x3
  (*h_filter)[0] = 1.;    (*h_filter)[1] = 1.;    (*h_filter)[2] = 1.;
  (*h_filter)[3] = 1.;    (*h_filter)[4] = 1.;    (*h_filter)[5] = 1.;
  (*h_filter)[6] = 1.;    (*h_filter)[7] = 1.;    (*h_filter)[8] = 1.;

  for (int i = 0; i < 9; i++)
    (*h_filter)[i] /= 9.0;

#elif FILTER == 5
  //Blur3x3
  (*h_filter)[0] = 1.;    (*h_filter)[1] = 2.;    (*h_filter)[2] = 1.;
  (*h_filter)[3] = 2.;    (*h_filter)[4] = 4.;    (*h_filter)[5] = 2.;  
  (*h_filter)[6] = 1.;    (*h_filter)[7] = 2.;    (*h_filter)[8] = 1.;

  for (int i = 0; i < 9; i++)
    (*h_filter)[i] /= 16.0;

#elif FILTER == 6
  //Blur5x5
  (*h_filter)[0] = 1.;   (*h_filter)[1] = 1.;   (*h_filter)[2] = 1.;    (*h_filter)[3] = 1.;    (*h_filter)[4] = 1.;
  (*h_filter)[5] = 1.;   (*h_filter)[6] = 4.;   (*h_filter)[7] = 4.;    (*h_filter)[8] = 4.;    (*h_filter)[9] = 1.;
  (*h_filter)[10] = 1.;  (*h_filter)[11] = 4.;  (*h_filter)[12] = 12.;  (*h_filter)[13] = 4.;   (*h_filter)[14] = 1.;
  (*h_filter)[15] = 1.;  (*h_filter)[16] = 4.;  (*h_filter)[17] = 4.;   (*h_filter)[18] = 4.;   (*h_filter)[19] = 1.;
  (*h_filter)[20] = 1.;  (*h_filter)[21] = 1.;  (*h_filter)[22] = 1.;   (*h_filter)[23] = 1.;   (*h_filter)[24] = 1.;

  for (int i = 0; i < 25; i++)
    (*h_filter)[i] /= 25.0;

#elif FILTER == 7

  //GaussNxN; N = GAUSSIAN_SZ
  const float KernelSigma = 2.;

  float filterSum = 0.f; //for normalization

  for (int r = -KernelWidth / 2; r <= KernelWidth / 2; ++r) {
    for (int c = -KernelWidth / 2; c <= KernelWidth / 2; ++c) {
      float filterValue = expf(-(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
      (*h_filter)[(r + KernelWidth / 2) * KernelWidth + c + KernelWidth / 2] = filterValue;
      filterSum += filterValue;
    }
  }

  float normalizationFactor = 1.f / filterSum;

  for (int r = -KernelWidth / 2; r <= KernelWidth / 2; ++r) {
    for (int c = -KernelWidth / 2; c <= KernelWidth / 2; ++c) {
      (*h_filter)[(r + KernelWidth / 2) * KernelWidth + c + KernelWidth / 2] *= normalizationFactor;
    }
  }

#elif FILTER == 8
  //SobelHorizontal3x3
  (*h_filter)[0] = -1.;   (*h_filter)[1] = -2.;   (*h_filter)[2] = -1.;
  (*h_filter)[3] = 0;     (*h_filter)[4] = 0;     (*h_filter)[5] = 0;
  (*h_filter)[6] = 1.;    (*h_filter)[7] = 2.;    (*h_filter)[8] = 1.;

#elif FILTER == 9
  //SobelVertical3x3
  (*h_filter)[0] = -1.;   (*h_filter)[1] = 0;     (*h_filter)[2] = 1.;
  (*h_filter)[3] = -2.;   (*h_filter)[4] = 0;     (*h_filter)[5] = 2.;
  (*h_filter)[6] = -1.;   (*h_filter)[7] = 0;     (*h_filter)[8] = 1.;

#else
  //Laplaciano 5x5
  (*h_filter)[0] = 0;     (*h_filter)[1] = 0;    (*h_filter)[2] = -1.;  (*h_filter)[3] = 0;    (*h_filter)[4] = 0;
  (*h_filter)[5] = 1.;    (*h_filter)[6] = -1.;  (*h_filter)[7] = -2.;  (*h_filter)[8] = -1.;  (*h_filter)[9] = 0;
  (*h_filter)[10] = -1.;  (*h_filter)[11] = -2.; (*h_filter)[12] = 17.; (*h_filter)[13] = -2.; (*h_filter)[14] = -1.;
  (*h_filter)[15] = 1.;   (*h_filter)[16] = -1.; (*h_filter)[17] = -2.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0;
  (*h_filter)[20] = 1.;   (*h_filter)[21] = 0;   (*h_filter)[22] = -1.; (*h_filter)[23] = 0;   (*h_filter)[24] = 0;

#endif
}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redFiltered, 
                        unsigned char *d_greenFiltered, 
                        unsigned char *d_blueFiltered,
                        const int filterWidth)
{
  //TODO: Calcular tama�os de bloque
  const dim3 blockSize = {BLOCK_SZ, BLOCK_SZ, 1};
  const dim3 gridSize = { ((unsigned int)numCols-1)/blockSize.x+1, ((unsigned int)numRows-1)/blockSize.y+1, 1 };

  //TODO: Lanzar kernel para separar imagenes RGBA en diferentes colores
  separateChannels <<<gridSize, blockSize >>> (d_inputImageRGBA,
    numRows,
    numCols,
    d_red,
    d_green,
    d_blue
    );

  //TODO: Ejecutar convoluci�n. Una por canal

#if SHARED == 1

  box_filter_shared <<<gridSize, blockSize, sizeof(unsigned char) * (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) >>> (
    d_red,
    d_redFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter_shared <<<gridSize, blockSize, sizeof(unsigned char) * (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) >>> (
    d_green,
    d_greenFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter_shared <<<gridSize, blockSize, sizeof(unsigned char) * (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) >>> (
    d_blue,
    d_blueFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );


#else
  box_filter<<<gridSize, blockSize >>> (d_red,
    d_redFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter << <gridSize, blockSize >> > (d_green,
    d_greenFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter << <gridSize, blockSize >> > (d_blue,
    d_blueFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

#endif

  // Recombining the results. 
  recombineChannels<<<gridSize, blockSize>>>(d_redFiltered,
                                             d_greenFiltered,
                                             d_blueFiltered,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}
