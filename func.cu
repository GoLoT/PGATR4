//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

//Definimos tama�o de bloque en preprocesador para facilidad al hacer pruebas
#define BLOCK_SZ 32
//Definimos en preprocesador para poder inicializar array de memoria constante
#define KERNEL_SZ 5
__constant__ float d_filterConst[KERNEL_SZ*KERNEL_SZ];
//Definimos para facilitar el cambio entre los kernels de memoria compartida y global
#define SHARED 1

__global__
void box_filter_shared(const unsigned char* const inputChannel,
  unsigned char* const outputChannel,
  int numRows, int numCols,
  const float* const filter, const int filterWidth)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  // NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
  // los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.

  extern __shared__ unsigned char image_shared[];

  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
    blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  //Shared size siempre deber�a ser par, ya que blockdim.x, blockdim.y
  //y filterwidth-1 siempre deber�n ser pares
  const int sharedSize = (blockDim.x + filterWidth - 1) * (blockDim.y + filterWidth - 1);
  const int halfSharedSize = sharedSize / 2;
  const int halfFilterWidth = filterWidth / 2;
  const int threadNum = threadIdx.y * blockDim.x + threadIdx.x;
  const int width = blockDim.x + filterWidth - 1;

  if(threadNum < halfSharedSize)
  {
    //Calculamos coordenadas de imagen de la secci�n a mapear en shared memory
    const int startX = blockIdx.x * blockDim.x - halfFilterWidth;
    const int startY = blockIdx.y * blockDim.y - halfFilterWidth;

    //Calculamos las coordenadas en shared memory
    int sharedY = threadNum / width;
    int sharedX = threadNum - sharedY * width;
    //Pasamos a coordenadas de imagen
    sharedX += startX;
    sharedY += startY;
    //Hacemos clamp para asegurar que no nos salimos de la imagen
    sharedY = sharedY >= numRows ? numRows - 1 : sharedY < 0 ? 0 : sharedY;
    sharedX = sharedX >= numCols ? numCols - 1 : sharedX < 0 ? 0 : sharedX;

    image_shared[threadNum] = inputChannel[sharedY * width + sharedX];

    sharedY = (threadNum + halfSharedSize) / width;
    sharedX = (threadNum + halfSharedSize) - sharedY * width;
    sharedX += startX;
    sharedY += startY;
    sharedY = sharedY >= numRows ? numRows - 1 : sharedY < 0 ? 0 : sharedY;
    sharedX = sharedX >= numCols ? numCols - 1 : sharedX < 0 ? 0 : sharedX;

    image_shared[threadNum + halfSharedSize] = inputChannel[sharedY * width + sharedX];

  }

  __syncthreads();

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;
  int filterRadius = filterWidth / 2;
  /*float result = 0;
  for (int j = -filterRadius; j <= filterRadius; j++)
    for (int i = -filterRadius; i <= filterRadius; i++) {
      int x = threadIdx.x + halfFilterWidth + i;
      int y = threadIdx.y + halfFilterWidth + j;

      float f = d_filterConst[(j + filterRadius)*filterWidth + i + filterRadius];
      float c = image_shared[y*width + x];
      result += f * c;
      //result += (float)d_filterConst[(j + filterRadius)*filterWidth + i + filterRadius] * (float)image_shared[y*width + x];
    }
  outputChannel[thread_1D_pos] = result > 255 ? 255 : result < 0 ? 0 : (char)result;*/
  outputChannel[thread_1D_pos] = image_shared[(threadIdx.y + halfFilterWidth)*width + threadIdx.x + halfFilterWidth];
}

__global__
void box_filter(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  // NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
  // los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.

  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
    blockIdx.y * blockDim.y + threadIdx.y);
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;
  int filterRadius = filterWidth / 2;
  float result = 0;
  for (int j = -filterRadius; j <= filterRadius; j++)
    for (int i = -filterRadius; i <= filterRadius; i++)  {
      int x = thread_2D_pos.x + i;
      x = x >= numCols ? numCols - 1 : x;
      x = x < 0 ? 0 : x;
      int y = thread_2D_pos.y + j;
      y = y >= numRows ? numRows - 1 : y;
      y = y < 0 ? 0 : y;
      //Sin memoria de constantes
      //result += (float) filter[(j + filterRadius)*filterWidth + i + filterRadius] * (float) inputChannel[y*numCols + x];
      //Con memoria de constantes
      result += (float)d_filterConst[(j + filterRadius)*filterWidth + i + filterRadius] * (float)inputChannel[y*numCols + x];
    }
  outputChannel[thread_2D_pos.y * numCols + thread_2D_pos.x] = result>255?255:result<0?0:(char)result;
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Reservar memoria para el filtro en GPU: d_filter, la cual ya esta declarada
  // Copiar el filtro  (h_filter) a memoria global de la GPU (d_filter)
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_filterConst), h_filter, sizeof(float) * filterWidth * filterWidth, 0, hipMemcpyHostToDevice));
}


void create_filter(float **h_filter, int *filterWidth){

  const int KernelWidth = KERNEL_SZ; //OJO CON EL TAMA�O DEL FILTRO//
  *filterWidth = KernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[KernelWidth * KernelWidth];
  
  /*
  //Filtro gaussiano: blur
  const float KernelSigma = 2.;

  float filterSum = 0.f; //for normalization

  for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
    for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
      float filterValue = expf( -(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
      (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] = filterValue;
      filterSum += filterValue;
    }
  }

  float normalizationFactor = 1.f / filterSum;

  for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
    for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
      (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] *= normalizationFactor;
    }
  }
  */

  //Laplaciano 5x5
  (*h_filter)[0] = 0;   (*h_filter)[1] = 0;    (*h_filter)[2] = -1.;  (*h_filter)[3] = 0;    (*h_filter)[4] = 0;
  (*h_filter)[5] = 1.;  (*h_filter)[6] = -1.;  (*h_filter)[7] = -2.;  (*h_filter)[8] = -1.;  (*h_filter)[9] = 0;
  (*h_filter)[10] = -1.;(*h_filter)[11] = -2.; (*h_filter)[12] = 17.; (*h_filter)[13] = -2.; (*h_filter)[14] = -1.;
  (*h_filter)[15] = 1.; (*h_filter)[16] = -1.; (*h_filter)[17] = -2.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0;
  (*h_filter)[20] = 1.;  (*h_filter)[21] = 0;   (*h_filter)[22] = -1.; (*h_filter)[23] = 0;   (*h_filter)[24] = 0;
  
  //TODO: crear los filtros segun necesidad
  //NOTA: cuidado al establecer el tama�o del filtro a utilizar

}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redFiltered, 
                        unsigned char *d_greenFiltered, 
                        unsigned char *d_blueFiltered,
                        const int filterWidth)
{
  //TODO: Calcular tama�os de bloque
  const dim3 blockSize = {BLOCK_SZ, BLOCK_SZ, 1};
  const dim3 gridSize = { ((unsigned int)numCols-1)/blockSize.x+1, ((unsigned int)numRows-1)/blockSize.y+1, 1 };

  //TODO: Lanzar kernel para separar imagenes RGBA en diferentes colores
  separateChannels <<<gridSize, blockSize >>> (d_inputImageRGBA,
    numRows,
    numCols,
    d_red,
    d_green,
    d_blue
    );

  //TODO: Ejecutar convoluci�n. Una por canal

#ifdef SHARED

  box_filter_shared <<<gridSize, blockSize, sizeof(unsigned char) * (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) >>> (
    d_red,
    d_redFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter_shared <<<gridSize, blockSize, sizeof(unsigned char) * (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) >>> (
    d_green,
    d_greenFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter_shared <<<gridSize, blockSize, sizeof(unsigned char) * (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) >>> (
    d_blue,
    d_blueFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );


#else
  box_filter<<<gridSize, blockSize >>> (d_red,
    d_redFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter << <gridSize, blockSize >> > (d_green,
    d_greenFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

  box_filter << <gridSize, blockSize >> > (d_blue,
    d_blueFiltered,
    numRows,
    numCols,
    d_filter,
    filterWidth
    );

#endif

  // Recombining the results. 
  recombineChannels<<<gridSize, blockSize>>>(d_redFiltered,
                                             d_greenFiltered,
                                             d_blueFiltered,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}
